/*
Single Author info:
nphabia Niklesh Ashok Phabiani
Group info:
nphabia Niklesh Ashok Phabiani
anjain2 Akshay Narendra Jain
rtnaik	Rohit Naik
*/

/* Program to compute Pi using Monte Carlo methods */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#define SEED 35791246

//Global function to calculate the value of pi
__global__ void calculate_value_of_pi(int *count, hiprandState *curandStates) {
	unsigned int thread_id = threadIdx.x + blockDim.x * blockIdx.x;
	hiprand_init(SEED, thread_id, 0, &curandStates[thread_id]);
	double x = hiprand_uniform(&curandStates[thread_id]);
      	double y = hiprand_uniform(&curandStates[thread_id]);
      	double z = x*x+y*y;
	if(z <= 1) {
		count[thread_id]++;
	}
}

int main(int argc, char** argv)
{
	int niter=0;
   	//double x,y;
   	int i,count=0; /* # of points in the 1st quadrant of unit circle */
   	//double z;
   	double pi;

	//number of iterations
   	niter = atoi(argv[1]);

	//random from library
	hiprandState *curandStates;

	//count devices and hosts
	int *count_d;
	int *count_h = (int*) malloc(niter * sizeof(int));  
	for(i = 0; i < niter; i++) {
		count_h[i] = 0;
	}

   	count=0;

	//Memory allocation
	hipMalloc((void**)&count_d, niter * sizeof(int));
	hipMalloc((void**)&curandStates, niter * sizeof(hiprandState));
	hipMemcpy(count_d, count_h, niter * sizeof(int), hipMemcpyHostToDevice);
	calculate_value_of_pi<<<10, niter/10>>>(count_d, curandStates);

	hipMemcpy(count_h, count_d, niter * sizeof(int), hipMemcpyDeviceToHost);

	for(i = 0; i < niter; i++) {
		count += count_h[i];
		//printf("Count: %d\n", count);
	}
	//Pi calculation
   	pi=(double)count/niter*4;
	hipFree(count_d);
	hipFree(curandStates);
   	printf("# of trials= %d , estimate of pi is %.16f \n",niter,pi);
}